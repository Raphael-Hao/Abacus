#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file np_elemwise_unary_op_basic.cu
 * \brief GPU Implementation of numpy unary functions.
 */
#include "../tensor/elemwise_binary_op.h"

namespace mxnet {
namespace op {

NNVM_REGISTER_OP(_npx_relu)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::relu>);

NNVM_REGISTER_OP(_npx_sigmoid)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::sigmoid>);

NNVM_REGISTER_OP(_np_copy)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::IdentityCompute<gpu>);

#define MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(__name$, __kernel$)       \
  NNVM_REGISTER_OP(__name$)                                               \
  .set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, __kernel$>)

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_negative, mshadow_op::negation);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_reciprocal, mshadow_op::reciprocal);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_absolute, mshadow_op::abs);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_sign, mshadow_op::sign);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_rint, mshadow_op::rint);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_ceil, mshadow_op::ceil);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_floor, mshadow_op::floor);

NNVM_REGISTER_OP(_npi_bitwise_not)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::ComputeInt<gpu, mshadow_op::bitwise_not>);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_trunc, mshadow_op::trunc);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_fix, mshadow_op::fix);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_square, mshadow_op::square);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_sqrt, mshadow_op::square_root);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_cbrt, mshadow_op::cube_root);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_exp, mshadow_op::exp);

NNVM_REGISTER_OP(_npi_log)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::Compute<gpu, mshadow_op::log>);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_log10, mshadow_op::log10);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_log2, mshadow_op::log2);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_log1p, mshadow_op::log1p);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_expm1, mshadow_op::expm1);

NNVM_REGISTER_OP(_npi_logical_not)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::ComputeLogic<gpu, mshadow_op::np_logical_not>);

NNVM_REGISTER_OP(_npi_isnan)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::ComputeLogic<gpu, mshadow_op::isnan>);

NNVM_REGISTER_OP(_npi_isinf)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::ComputeLogic<gpu, mshadow_op::isinf>);

NNVM_REGISTER_OP(_npi_isposinf)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::ComputeLogic<gpu, mshadow_op::isposinf>);

NNVM_REGISTER_OP(_npi_isneginf)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::ComputeLogic<gpu, mshadow_op::isneginf>);

NNVM_REGISTER_OP(_npi_isfinite)
.set_attr<FCompute>("FCompute<gpu>", UnaryOp::ComputeLogic<gpu, mshadow_op::isfinite>);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_sin, mshadow_op::sin);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_cos, mshadow_op::cos);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_tan, mshadow_op::tan);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_arcsin, mshadow_op::arcsin);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_arccos, mshadow_op::arccos);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_arctan, mshadow_op::arctan);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_degrees, mshadow_op::degrees);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_radians, mshadow_op::radians);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_sinh, mshadow_op::sinh);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_cosh, mshadow_op::cosh);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_tanh, mshadow_op::tanh);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_arcsinh, mshadow_op::arcsinh);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_arccosh, mshadow_op::arccosh);

MXNET_OPERATOR_REGISTER_NUMPY_UNARY_GPU(_npi_arctanh, mshadow_op::arctanh);

NNVM_REGISTER_OP(_npi_around)
.set_attr<FCompute>("FCompute<gpu>", AroundOpForward<gpu>);

NNVM_REGISTER_OP(_npi_nan_to_num)
.set_attr<FCompute>("FCompute<gpu>", NumpyNanToNumOpForward<gpu>);

NNVM_REGISTER_OP(_npi_backward_nan_to_num)
.set_attr<FCompute>("FCompute<gpu>", NumpyNanToNumOpBackward<gpu>);

}  // namespace op
}  // namespace mxnet
